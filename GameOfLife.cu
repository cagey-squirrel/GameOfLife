
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define for_x for (int x = 0; x < w; x++)
#define for_y for (int y = 0; y < h; y++)
#define for_xy for_x for_y

#define NUM_OF_GPU_THREADS 1024
#define BLOCK_DIM_x 32
#define BLOCK_DIM_y 32

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void print_matrix(unsigned *u, int h, int w) {
    for(int i = 0; i < h+2; i++) {
        for(int j = 0; j < w+2; j++) {
            printf("%d ", u[i*(w+2) + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void init(unsigned *u, int w, int h) {
    for_xy u[y*w + x] = rand() < RAND_MAX / 10 ? 1 : 0;
}

void show(unsigned *u, int w, int h) {
    printf("\033[H");
    for_y {
        for_x printf(u[y*w + x] ? "\033[07m  \033[m" : "  ");
        printf("\033[E");
    }
    fflush(stdout);
}

void my_init(unsigned *u, int w, int h) {
    for(int y = 1; y < h+1; y++)
        for(int x = 1; x < w+1; x++) 
            u[y*w + x] = rand() < RAND_MAX / 2 ? 1 : 0;
}

void my_init_twice(unsigned *u1, unsigned* u2, int w, int h) {
    // Changing the order of loops to get more cache hits
    for(int y = 1; y < h-1; y++)
        for(int x = 1; x < w-1; x++) {
            u1[y*w + x] = rand() < RAND_MAX / 2 ? 1 : 0;
            u2[y*w + x] = u1[y*w + x];
        }
}

void my_evolve(unsigned **u, unsigned **new_p, int w, int h) {
    unsigned *univ = *u;
    unsigned *temp = *new_p;

    for(int y = 1; y < h - 1; y++) {
        for(int x = 1; x < w - 1; x ++) {
            unsigned n = univ[(y-1)*w + x-1] + univ[(y-1)*w + x] + univ[(y-1)*w + x+1] + univ[(y)*w + x-1] + univ[(y)*w + x+1] + univ[(y+1)*w + x-1] + univ[(y+1)*w + x] + univ[(y+1)*w + x+1];
            temp[y*w + x] = (n == 3 || (n == 2 && univ[y*w + x]));
        }
    }
    unsigned* t = *u;
    *u = *new_p;
    *new_p = t;
    //print_matrix(*u,w,h);
}

void my_evolve_parallel(unsigned **u, unsigned **new_p, int w, int h) {
    unsigned *univ = *u;
    unsigned *temp = *new_p;

    for(int y = 1; y < h - 1; y++) {
        for(int x = 1; x < w - 1; x ++) {
            unsigned n = univ[(y-1)*w + x-1] + univ[(y-1)*w + x] + univ[(y-1)*w + x+1] + univ[(y)*w + x-1] + univ[(y)*w + x+1] + univ[(y+1)*w + x-1] + univ[(y+1)*w + x] + univ[(y+1)*w + x+1];
            temp[y*w + x] = (n == 3 || (n == 2 && univ[y*w + x]));
        }
    }

    unsigned* t = *u;
    *u = *new_p;
    *new_p = t;
    //print_matrix(*u,w,h);
}

void evolve(unsigned *u, int w, int h) {

    unsigned* tem = (unsigned*) malloc(w*h*sizeof(unsigned));
    for_y for_x {
        int n = 0;
        for (int y1 = y - 1; y1 <= y + 1; y1++)
            for (int x1 = x - 1; x1 <= x + 1; x1++)
                if (u[((y1 + h) % h)*h + ((x1 + w) % w)]) n++;

        if (u[y*h+x]) n--;
        tem[y*h+x] = (n == 3 || (n == 2 && u[y*h+x]));
    }
    for_y for_x u[y*h+x] = 1;
    free(tem);
}

void game(unsigned *u, int w, int h, int iter) {
    for (int i = 0; i < iter; i++) {
#ifdef LIFE_VISUAL
        show(u, w, h);
#endif
        evolve(u, w, h);
#ifdef LIFE_VISUAL
        usleep(200000);
#endif
    }
}


void my_game(unsigned **u, unsigned **new_p, int w, int h, int iter) {
    for (int i = 0; i < iter; i++) {
#ifdef LIFE_VISUAL
        show(u, w, h);
#endif
        my_evolve(u, new_p, w+2, h+2);
#ifdef LIFE_VISUAL
        usleep(200000);
#endif
    }
}

__global__ void GameOfLifeKernel(unsigned *u, unsigned *t, int w, int h) {
    int index_x = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
    int index_y = blockIdx.y * (blockDim.y - 2) + threadIdx.y;
    int shared_index_x = threadIdx.x;
    int shared_index_y = threadIdx.y;
    int middle_square_pos = index_y * w + index_x;

    __shared__ int shared_u[BLOCK_DIM_y][BLOCK_DIM_x];

    if((index_x) < (w) && index_y < (h)) {
        shared_u[threadIdx.y][threadIdx.x] = u[middle_square_pos];
    }

    __syncthreads();

    if((index_x) < (w-1) && index_y < (h-1)) {
        if((shared_index_x > 0) && (shared_index_x < (blockDim.x - 1)) && (shared_index_y > 0) && (shared_index_y < (blockDim.y - 1))) {
            unsigned n = shared_u[shared_index_y-1][shared_index_x-1] + shared_u[shared_index_y-1][shared_index_x] + shared_u[shared_index_y-1][shared_index_x+1] + shared_u[shared_index_y][shared_index_x-1] + shared_u[shared_index_y][shared_index_x+1] + shared_u[shared_index_y+1][shared_index_x-1] + shared_u[shared_index_y+1][shared_index_x] + shared_u[shared_index_y+1][shared_index_x+1];
            t[middle_square_pos] = (n == 3 || (n == 2 && shared_u[shared_index_y][shared_index_x]));
        }
        
    }
    
    __syncthreads();

}

void my_game_parallel(unsigned **u, unsigned **new_p, int w, int h, int iter) {

    unsigned *u_gpu, *t_gpu;
    int size, grid_height, grid_width;

    // Calculating block and grid dimensions:
    grid_height = (h+BLOCK_DIM_y-3) / (BLOCK_DIM_y-2);
    grid_width = (w+BLOCK_DIM_x-3) / (BLOCK_DIM_x-2);
    
    dim3 grid_dim(grid_height, grid_width);
    dim3 block_dim(BLOCK_DIM_y, BLOCK_DIM_x);
    
    // Allocating and copying matrix to GPU
    size = (w+2) * (h+2) * sizeof(unsigned);
    hipMalloc(&u_gpu, size);
    hipMemcpy(u_gpu, *u, size, hipMemcpyHostToDevice);
    hipMalloc(&t_gpu, size);

    hipEvent_t start = hipEvent_t();
	hipEvent_t stop = hipEvent_t();
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    // Kernel call
    for (int i = 0; i < iter; i++) {
        GameOfLifeKernel<<< grid_dim, block_dim >>>(u_gpu, t_gpu, w+2, h+2);
        hipStreamQuery(0);
        unsigned* t = u_gpu;
        u_gpu = t_gpu;
        t_gpu = t;
    }

	// Compute elapsed time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed = 0.f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Parallel implementation execution time = %f \n", elapsed);

    

    hipMemcpy(*u, u_gpu, size, hipMemcpyDeviceToHost);
    hipFree( u_gpu );
    hipFree( t_gpu );



}

void copy_initialization_to_gold_version(unsigned *u, unsigned *u_gold, int w, int h) {
    for(int i = 0; i < h; i++) {
        for(int j = 0; j < w; j++) {
            u_gold[i*w+j] = u[(i+1)*(w+2) + (j+1)];
        }
    }
}



__global__ void my_game_cuda(int* devA, int* devB, int* devC, int n){
	// Calculate index
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if( idx < n ) devC[idx] = devA[idx] + devB[idx];
}

void compare_matrices(unsigned *u, unsigned* u_parallel, int w, int h) {
    for(int y = 1; y < h - 1; y++) 
        for(int x = 1; x < w - 1; x++) {
            if(u[y*w + x] != u_parallel[y*w + x]) {
                printf("at y = %d, x = %d", y, x);
                printf("\n\n Test FAILED \n");
                exit(-1);
            }
        }
    
    printf("\n\n Test PASSED \n");
}

int main(int c, char *v[]) {
    int w = 0, h = 0, iter = 0;
    unsigned *u;
    unsigned *u_parallel;
    unsigned *temp;
    unsigned *u_gold;

    if (c > 1) w = atoi(v[1]);
    if (c > 2) h = atoi(v[2]);
    if (c > 3) iter = atoi(v[3]);
    if (w <= 0) w = 30;
    if (h <= 0) h = 30;
    if (iter <= 0) iter = 1000;

    u_gold = (unsigned *)calloc((w) * (h), sizeof(unsigned));
    u =     (unsigned *)calloc((w+2) * (h+2), sizeof(unsigned));
    u_parallel =     (unsigned *)calloc((w+2) * (h+2), sizeof(unsigned));

    my_init_twice(u, u_parallel, w+2, h+2);
    copy_initialization_to_gold_version(u, u_gold, w, h);
    

    hipEvent_t start = hipEvent_t();
	hipEvent_t stop = hipEvent_t();
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    game(u_gold, w, h, iter);

	// Compute elapsed time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed = 0.f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Gold implementation execution time = %f \n", elapsed);
    free(u_gold);

    temp = (unsigned *)calloc((w+2) * (h+2), sizeof(unsigned));
	hipEventRecord(start, 0);

    my_game(&u, &temp, w, h, iter);

	// Compute elapsed time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	elapsed = 0.f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Sequential improved implementation execution time = %f \n", elapsed);

    hipEventRecord(start, 0);

    // Core call
    my_game_parallel(&u_parallel, &temp, w, h, iter);

    // Compute elapsed time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	elapsed = 0.f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Full implementation execution time = %f \n", elapsed);

	// release events
	hipEventDestroy(start);
	hipEventDestroy(stop);

    free(temp);

    compare_matrices(u, u_parallel, w+2, h+2);

    free(u);
    free(u_parallel);
    

}
